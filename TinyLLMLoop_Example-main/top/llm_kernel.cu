#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_fp8.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <stdio.h>
#include <stdlib.h>
#include <vector>

#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4 *>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4 *>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2 *>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162 *>(&(value))[0])
#define LDST64BITS(value) (reinterpret_cast<float2 *>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4 *>(&(value))[0])

// FP16
// HGEMM naive: compute one c[i,j]
// element per threads, all row major
__global__ void hgemm_f16_kernel(half *a, half *b, half *c, int M, int N,
                                       int K) {
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  int m = blockIdx.y * blockDim.y + threadIdx.y;

  if (m < M && n < N) {
    half psum = 0.0;
#pragma unroll
    for (int k = 0; k < K; k++) {
      // m row in a matrix, n col in b matrix
      psum += a[m * K + k] * b[k * N + n];
    }
    c[m * N + n] = psum; // c[m,n]
  }
}

#include <torch/extension.h>
#include <torch/types.h>
#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func)                                   \
  m.def(STRINGFY(func), &func, STRINGFY(func));

#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                                   \
  if (((T).options().dtype() != (th_type))) {                                  \
    std::cout << "Tensor Info:" << (T).options() << std::endl;                 \
    throw std::runtime_error("values must be " #th_type);                      \
  }

#define CHECK_TORCH_TENSOR_SHAPE(T, S0, S1)                                    \
  if (((T).size(0) != (S0)) || ((T).size(1) != (S1))) {                        \
    throw std::runtime_error("Tensor size mismatch!");                         \
  }

// HGEMM naive: compute one c[i,j] element per threads, all row major
void hgemm_llm_kernel(torch::Tensor a, torch::Tensor b, torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kHalf)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1);
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)
  constexpr int BM = 32;
  constexpr intt BN = 32;

  dim3 block(BN, BM);
  dim3 grid((N + BN - 1) / BN, (M + BM - 1) / BM);

  hgemm_f16_kernel<<<grid, block>>>(
      reinterpret_cast<half *>(a.data_ptr()),
      reinterpret_cast<half *>(b.data_ptr()),
      reinterpret_cast<half *>(c.data_ptr()), M, N, K);
}
